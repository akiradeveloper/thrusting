#include "hip/hip_runtime.h"
#include <thrusting/tuple.h>
#include <thrusting/dtype/tuple/int.h>
#include <thrusting/iterator.h>
#include <thrusting/iterator/zip_iterator.h>

#include <iostream>

#include <gtest/gtest.h>

namespace {
  using namespace thrusting::op;
}

TEST(tuple, plus){
  // T + T
  EXPECT_EQ(thrusting::make_tuple2(3,5), thrusting::make_tuple2(2,3) + thrusting::make_tuple2(1,2));
} 

TEST(tuple, minus){
  // T - T
  EXPECT_EQ(thrusting::make_tuple2(1,1), thrusting::make_tuple2(2,3) - thrusting::make_tuple2(1,2));
}

TEST(tuple, multiplies){
  // v * T
  EXPECT_EQ(thrusting::make_tuple2(2,2), 2 * thrusting::make_tuple2(1,1));
}

TEST(tuple, divides){
  // T / v
  EXPECT_EQ(thrusting::make_tuple2(1,1), thrusting::make_tuple2(2,2) / 2);
}

TEST(tuple, make_string){
  EXPECT_EQ("(1, 2)", thrusting::make_string(thrusting::make_tuple2(1,2)));
}
  
TEST(tuple, ostream){
  std::cout << thrusting::make_tuple2(1,2) << std::endl;
}

TEST(tuple, arithmatic_priority){
  thrusting::int2 x = thrusting::make_tuple2(1,2);
  thrusting::int2 y = thrusting::make_tuple2(3,4);
  
  EXPECT_EQ(thrusting::make_tuple2(7, 10), x + 2 * y);
  EXPECT_EQ(thrusting::make_tuple2(8, 12), 2 * (x + y));
}
