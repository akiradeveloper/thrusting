#include "hip/hip_runtime.h"
#include <thrusting/tuple.h>
#include <thrusting/dtype/tuple/int.h>

#include <gtest/gtest.h>
#include <iostream>

namespace {
  using namespace thrusting::op;
}

TEST(Tuple, Plus){
  // T + T
  EXPECT_EQ(thrusting::make_tuple2(3,5), thrusting::make_tuple2(2,3) + thrusting::make_tuple2(1,2));
} 

TEST(Tuple, Minus){
  // T - T
  EXPECT_EQ(thrusting::make_tuple2(1,1), thrusting::make_tuple2(2,3) - thrusting::make_tuple2(1,2));
}

TEST(Tuple, Multiplies2){
  // v * T
  EXPECT_EQ(thrusting::make_tuple2(2,2), 2 * thrusting::make_tuple2(1,1));
}

TEST(Tuple, Divides){
  // T / v
  EXPECT_EQ(thrusting::make_tuple2(1,1), thrusting::make_tuple2(2,2) / 2);
}

TEST(Tuple, MakeString){
  EXPECT_EQ("(1, 2)", thrusting::make_string(thrusting::make_tuple2(1,2)));
}
  
TEST(Tuple, OStream){
  std::cout << thrusting::make_tuple2(1,2) << std::endl;
}

TEST(Tuple, ArithmaticOrdering){
  thrusting::int2 x = thrusting::make_tuple2(1,2);
  thrusting::int2 y = thrusting::make_tuple2(3,4);
  
  EXPECT_EQ(thrusting::make_tuple2(7, 10), x + 2 * y);
  EXPECT_EQ(thrusting::make_tuple2(8, 12), 2 * (x + y));
}
