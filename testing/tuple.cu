#include "hip/hip_runtime.h"
#include <thrusting/tuple.h>
#include <thrusting/dtype.h>

#include <gtest/gtest.h>
#include <iostream>

TEST(Tuple, Plus){
  // T + T
  EXPECT_EQ(thrusting::make_tuple2(3,5), thrusting::make_tuple2(2,3) + thrusting::make_tuple2(1,2));
} 

TEST(Tuple, Minus){
  // T - T
  EXPECT_EQ(thrusting::make_tuple2(1,1), thrusting::make_tuple2(2,3) - thrusting::make_tuple2(1,2));
}

TEST(Tuple, Multiplies){
  // T * T
  EXPECT_EQ(thrusting::make_tuple2(2,6), thrusting::make_tuple2(2,3)*thrusting::make_tuple2(1,2));
}

TEST(Tuple, Multiplies2){
  // T * v
  EXPECT_EQ(thrusting::make_tuple2(2,2), thrusting::make_tuple2(1,1) * 2);
}

TEST(Tuple, Multiplies3){
  // v * T
  EXPECT_EQ(thrusting::make_tuple2(2,2), 2 * thrusting::make_tuple2(1,1));
}

TEST(Tuple, Divides){
  // T / T
  EXPECT_EQ(thrusting::make_tuple2(1,2), thrusting::make_tuple2(4,2) / thrusting::make_tuple2(4,1));
}

TEST(Tuple, Divides2){
  // T / v
  EXPECT_EQ(thrusting::make_tuple2(1,1), thrusting::make_tuple2(2,2) / 2);
}
  
TEST(Tuple, OStream){
  std::cout << thrusting::make_tuple2(2,1) << std::endl;
}

TEST(Tuple, ArithmaticOrdering){
  thrusting::int2 x = thrusting::make_tuple2(1,2);
  thrusting::int2 y = thrusting::make_tuple2(3,4);
  thrusting::int2 z = thrusting::make_tuple2(5,6);
  
  EXPECT_EQ(thrusting::make_tuple2(16, 26), x + y * z);
  EXPECT_EQ(thrusting::make_tuple2(8, 20), x * (y+z));
}
