#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <thrusting/thrusting.h>
#include <iostream>

// namespace thrusting { 

// copied from tuple_operator.h
// and succeeded compilation why?
template<typename T0, typename T1>
std::ostream &operator<<(std::ostream &os, const thrust::tuple<T0, T1> &x){
  std::stringstream ss;
  ss << "(" << 
thrust::get<0>(x)
 << ", " << 
thrust::get<1>(x)
 << ")";
  os << ss.str();
  return os;
}
// } // end thrusting

TEST(TupleTest, Plus){
  using namespace thrusting;
  EXPECT_EQ(make_tuple2<int>(2,3) + make_tuple2<int>(1,2), make_tuple2<int>(3,5));
} 

TEST(Tuple, Print){
  using namespace thrusting;
  const thrusting::int2 const x = make_tuple2<int>(2,1);
  // std::cout << x << std::endl;
}
