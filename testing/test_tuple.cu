#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <thrusting/thrusting.h>
#include <iostream>

TEST(Tuple, Plus){
  thrust::int2 x = thrust::make_tuple2(2,3) + thrust::make_tuple2(1,2);
  EXPECT_EQ(thrust::make_tuple2<int>(3,5), x);
} 

TEST(Tuple, Multiplies){
  EXPECT_EQ(thrust::make_tuple2<int>(2,6), thrust::make_tuple2(2,3)*thrust::make_tuple2(1,2));
}

TEST(Tuple, Print){
  thrust::int2 x = thrust::make_tuple2<int>(2,1);
  // std::cout << x << std::endl;
}
