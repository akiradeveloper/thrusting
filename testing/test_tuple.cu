#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <thrust/device_vector.h>
#include <thrusting/thrusting.h>

TEST(Tuple, Plus){
  thrusting::int2 x = thrusting::make_tuple2(2,3) + thrusting::make_tuple2(1,2);
  EXPECT_EQ(thrusting::make_tuple2<int>(3,5), x);
} 

TEST(Tuple, Multiplies){
  EXPECT_EQ(thrusting::make_tuple2<int>(2,6), thrusting::make_tuple2(2,3)*thrusting::make_tuple2(1,2));
}

TEST(Tuple, Devides){
  EXPECT_EQ(thrusting::make_tuple2<int>(1,2), thrusting::make_tuple2(4,2)/thrusting::make_tuple2(4,1));
}

TEST(Tuple, Print){
  thrusting::int2 x = thrusting::make_tuple2<int>(2,1);
  // std::cout << x << std::endl;
}
