#include "hip/hip_runtime.h"
#include <thrusting/tuple.h>
#include <thrusting/dtype.h>

#include <gtest/gtest.h>
#include <thrust/device_vector.h>
#include <iostream>

using namespace thrusting::op::tuple;

TEST(Tuple, ArithmaticOrdering){
  thrusting::int2 x = thrusting::make_tuple2(1,2);
  thrusting::int2 y = thrusting::make_tuple2(3,4);
  thrusting::int2 z = thrusting::make_tuple2(5,6);
  
  EXPECT_EQ(thrusting::make_tuple2(16, 26), x + y * z);
  EXPECT_EQ(thrusting::make_tuple2(8, 20), x * (y+z));
}

TEST(Tuple, Plus){
  // T + T
  EXPECT_EQ(thrusting::make_tuple2<int>(3,5), thrusting::make_tuple2(2,3) + thrusting::make_tuple2(1,2));
} 

TEST(Tuple, Minus){
  // T - T
  EXPECT_EQ(thrusting::make_tuple2(1,1), thrusting::make_tuple2(2,3) - thrusting::make_tuple2(1,2));
}

TEST(Tuple, Multiplies){
  // T * T
  EXPECT_EQ(thrusting::make_tuple2<int>(2,6), thrusting::make_tuple2(2,3)*thrusting::make_tuple2(1,2));
}

TEST(Tuple, Multiplies2){
  // T * v
  EXPECT_EQ(thrusting::make_tuple2(2,2), thrusting::make_tuple2(1,1) * 2);
}

TEST(Tuple, Multiplies3){
  // v * T
  EXPECT_EQ(thrusting::make_tuple2(2,2), 2 * thrusting::make_tuple2(1,1));
}

TEST(Tuple, Divides){
  // T / T
  EXPECT_EQ(thrusting::make_tuple2<int>(1,2), thrusting::make_tuple2(4,2) / thrusting::make_tuple2(4,1));
}

TEST(Tuple, Divides2){
  // T / v
  EXPECT_EQ(thrusting::make_tuple2(1,1), thrusting::make_tuple2(2,2) / 2);
}
  
TEST(Tuple, OStream){
  std::cout << thrusting::make_tuple2<int>(2,1) << std::endl;
}
