#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <thrusting/thrusting.h>
#include <iostream>

// letting this function in std namespace and compilation scceeded, why?
//template<typename T0, typename T1>
//std::ostream &operator<<(std::ostream &os, const thrust::tuple<T0, T1> &x){
//  std::stringstream ss;
//  ss << "(" << 
//thrust::get<0>(x)
// << ", " << 
//thrust::get<1>(x)
// << ")";
//  os << ss.str();
//  return os;
//}

TEST(TupleTest, Plus){
  // ADL not effective to alias.
  // thrusting int2 aliased to thrust::tuple<int, int> and ADL failed.
  thrusting::int2 x = thrusting::make_tuple2(2,3) * thrusting::make_tuple2(1,2);
  EXPECT_EQ(x, thrusting::make_tuple2<int>(3,5));
} 

TEST(Tuple, Print){
  thrusting::int2 x = thrust::make_tuple2<int>(2,1);
  // std::cout << x << std::endl;
}
