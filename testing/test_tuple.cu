#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <thrusting/thrusting.h>
#include <iostream>

TEST(TupleTest, Plus){
  thrust::int2 x = thrust::make_tuple2(2,3) * thrust::make_tuple2(1,2);
  EXPECT_EQ(x, thrust::make_tuple2<int>(3,5));
} 

TEST(Tuple, Print){
  thrust::int2 x = thrust::make_tuple2<int>(2,1);
  // std::cout << x << std::endl;
}
