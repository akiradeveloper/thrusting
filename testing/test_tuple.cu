#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <thrusting/thrusting.h>
#include <iostream>

TEST(Tuple, Plus){
  thrust::int2 x = thrust::make_tuple2(2,3) + thrust::make_tuple2(1,2);
  EXPECT_EQ(thrust::make_tuple2<int>(3,5), x);
} 

TEST(Tuple, Multiplies){
  EXPECT_EQ(thrust::make_tuple2<int>(2,6), thrust::make_tuple2(2,3)*thrust::make_tuple2(1,2));
}

TEST(Tuple, Devides){
  EXPECT_EQ(thrust::make_tuple<int>(1,2), thrust::make_tuple2(4,2) / thrust::make_tuple(4,1));
}

TEST(Tuple, Print){
  thrust::int2 x = thrust::make_tuple2<int>(2,1);
  // std::cout << x << std::endl;
}
