#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <thrusting/thrusting.h>
#include <iostream>

// letting this function in std namespace and compilation scceeded, why?
//template<typename T0, typename T1>
//std::ostream &operator<<(std::ostream &os, const thrust::tuple<T0, T1> &x){
//  std::stringstream ss;
//  ss << "(" << 
//thrust::get<0>(x)
// << ", " << 
//thrust::get<1>(x)
// << ")";
//  os << ss.str();
//  return os;
//}

TEST(TupleTest, Plus){
  using namespace thrusting;
  EXPECT_EQ(make_tuple2<int>(2,3) + make_tuple2<int>(1,2), make_tuple2<int>(3,5));
} 

TEST(Tuple, Print){
  using namespace thrusting;
  const thrusting::int2 const x = make_tuple2<int>(2,1);
  // std::cout << x << std::endl;
}
