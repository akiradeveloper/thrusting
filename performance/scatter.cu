#include <thrusting/vector.h>
#include <thrusting/algorithm/scatter.h>
#include <thrusting/real.h>
#include <thrusting/time.h>

#include <thrust/sequence.h>
#include <thrust/reverse.h>

#include <algorithm>

namespace {
  using namespace thrusting;
}

int main(void){
  size_t N = 1000000;

  thrust::host_vector<int> _value(N);
  thrust::sequence(_value.begin(), _value.end());

  thrust::host_vector<int> _idx(N);
  thrust::sequence(_idx.begin(), _idx.end());
  
  thrusting::stopwatch sw("thrusting"); 

  for(int i=0; i<3; ++i){
    std::random_shuffle(_value.begin(), _value.end());
    vector<int>::type value(_value.begin(), _value.end());

    std::random_shuffle(_idx.begin(), _idx.end());
    vector<int>::type idx(_idx.begin(), _idx.end());

    sw.begin();
    thrusting::scatter(
      value.begin(),
      value.end(),
      idx.begin(),
      value.begin()); // in-place
    
    sw.end();
  }
   
  sw.show();

  thrusting::stopwatch sw2("thrust");     
  vector<int>::type output(N);     
  
  for(int i=0; i<10; ++i){
    std::random_shuffle(_value.begin(), _value.end());
    vector<int>::type value(_value.begin(), _value.end());

    std::random_shuffle(_idx.begin(), _idx.end());
    vector<int>::type idx(_idx.begin(), _idx.end());

    sw2.begin();
    thrust::scatter(
      value.begin(),
      value.end(),
      idx.begin(),
      output.begin());

    // not needed though
    hipEvent_t stop;
    hipEventCreate( &stop );
    hipEventSynchronize( stop );

    sw2.end();
  }
  
  sw2.show();
  
  return 0;
}
