#include <thrusting/vector.h>
#include <thrusting/algorithm/scatter.h>
#include <thrusting/real.h>
#include <thrusting/time.h>
#include <thrusting/timer.h>

#include <thrust/sequence.h>

#include <algorithm>

namespace {
  using namespace thrusting;
}

int main(void){
  size_t N = 10000000;

  thrust::host_vector<int> _value(N);
  thrust::sequence(_value.begin(), _value.end());

  thrust::host_vector<int> _idx(N);
  thrust::sequence(_idx.begin(), _idx.end());
  
  thrusting::stopwatch sw("thrusting"); 

  for(int i=0; i<1; ++i){
    std::random_shuffle(_value.begin(), _value.end());
    vector<int>::type value(_value.begin(), _value.end());

    std::random_shuffle(_idx.begin(), _idx.end());
    vector<int>::type idx(_idx.begin(), _idx.end());

    hipDeviceSynchronize();
    sw.begin();

    thrusting::scatter(
      value.begin(),
      value.end(),
      idx.begin(),
      value.begin()); // in-place
    
    hipDeviceSynchronize();
    sw.end();
  }
   
  sw.show();

  thrusting::stopwatch sw2("thrust");     
  vector<int>::type output(N);     
  
  for(int i=0; i<1; ++i){
    std::random_shuffle(_value.begin(), _value.end());
    vector<int>::type value(_value.begin(), _value.end());

    std::random_shuffle(_idx.begin(), _idx.end());
    vector<int>::type idx(_idx.begin(), _idx.end());

    hipDeviceSynchronize();
    sw2.begin();
    thrust::scatter(
      value.begin(),
      value.end(),
      idx.begin(),
      output.begin());

    hipDeviceSynchronize();
    sw2.end();
  }
  
  sw2.show();
  
  return 0;
}
