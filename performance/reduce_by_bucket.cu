#include "hip/hip_runtime.h"
#include <thrusting/algorithm/reduce_by_bucket.h>
#include <thrusting/vector.h>
#include <thrusting/time.h>
#include <thrusting/list.h>

#include <thrust/iterator/constant_iterator.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#include <stdlib.h>

#include <algorithm>

namespace {
  using namespace thrusting;
}

namespace thrusting {
struct random {
  random(int seed)
  {
    srand(seed);
  }
  int operator()(int ceil) const {
    float r = rand();
    float n = ceil * (r-1) / RAND_MAX;
    return (int) n;
  }  
};
} // END thrusting

int main(void){
  size_t N_VALUE = 10000000;
  size_t N_BUCKET = 500000;

  vector<int>::type prefix_output(N_BUCKET); 
  vector<int>::type cnt_output(N_BUCKET);
  vector<int>::type value_output(N_BUCKET);

  thrusting::stopwatch sw("reduce_by_bucket"); 

  for(int i=0; i<3; ++i){
    thrust::host_vector<int> _value(N_VALUE);
    thrust::sequence(_value.begin(), _value.end());
    std::random_shuffle(_value.begin(), _value.end());
    vector<int>::type value(_value.begin(), _value.end());

    thrust::host_vector<int> _idx(N_VALUE);
    int lucky_seed = 777;
    thrusting::random ran(lucky_seed);
    thrust::transform(
      thrust::make_constant_iterator(N_BUCKET),
      thrust::make_constant_iterator(N_BUCKET) + N_VALUE,
      _idx.begin(),
      ran); 
    vector<int>::type idx(_idx.begin(), _idx.end());
    thrust::sort(idx.begin(), idx.end());

    hipDeviceSynchronize();
    sw.begin();

    thrusting::reduce_by_bucket(
      N_VALUE,
      value.begin(),
      idx.begin(),
      N_BUCKET,
      prefix_output.begin(),
      cnt_output.begin(),
      value_output.begin(),
      777); // NULL VALUE
    
    hipDeviceSynchronize();
    sw.end();
  }
  
  sw.show();
  
  return 0;
}
