/*
  Copied From Thrust Library v-1.3.0
  and refactored a little bit. 
  
  Akira Hayakawa, 2010
*/


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime_api.h>

void usage(const char *name){
  printf("usage: %s [device_id]\n", name);
}

int main(int argc, char **argv){
  int num_devices = 0;
  int device_id = 0;

  if(argc != 2){
    usage(argv[0]);
    exit(-1);
  }

  device_id = atoi(argv[1]);

  hipGetDeviceCount(&num_devices);

  if(device_id >= num_devices){
    printf("No available device with id %d\n", device_id);
    return -1;
  } 
   
  hipDeviceProp_t properties;
  hipGetDeviceProperties(&properties, device_id);
  printf("sm_%d%d", properties.major, properties.minor);
  return 0;
}
